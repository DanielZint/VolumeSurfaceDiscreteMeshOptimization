#include "hip/hip_runtime.h"
#include "DMOHex.h"

#include "hip/hip_runtime.h"
#include ""
#include <thrust/count.h>
#include <thrust/sequence.h>
#include "CudaUtil.h"
#include "CudaAtomic.h"
#include "DMOCommon.h"
#include "SurfaceConfig.h"

#include "io/FileWriter.h"

#include "DMOHexImplCub.h"

//using namespace SurfLS;
//using namespace Surf1D;

#ifdef USECUB
using namespace DMOImplCub;
#else
using namespace DMOImplAtomic;
#endif


namespace DMO {



	__global__ static void k_qualityHistogram(MeshHexDevice* mesh, ArrayView<int> q_vec, ArrayView<float> q_min, int n_cols, QualityCriterium q_crit) {
		for (int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < mesh->nHexahedra; idx += blockDim.x * gridDim.x) {
			const Hexahedron& tet = mesh->hexahedra[idx];
			const Vec3f points[8] = { mesh->vertexPoints[tet.v0], mesh->vertexPoints[tet.v1], mesh->vertexPoints[tet.v2], mesh->vertexPoints[tet.v3],
				mesh->vertexPoints[tet.v4], mesh->vertexPoints[tet.v5], mesh->vertexPoints[tet.v6], mesh->vertexPoints[tet.v7] };
			float q = qualityHex(points, q_crit);
			myAtomicMin(&q_min[0], q);

			q = fminf(0.9999f, q);
			size_t index = size_t(q * n_cols);
			atomicAdd(&q_vec[index], 1);
		}
	}

	// finds the minimum element quality
	__global__ static void k_findMinimumQuality(MeshHexDevice* mesh, ArrayView<float> q_min, QualityCriterium q_crit) {
		for (int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < mesh->nHexahedra; idx += blockDim.x * gridDim.x) {
			const Hexahedron& tet = mesh->hexahedra[idx];
			const Vec3f points[8] = { mesh->vertexPoints[tet.v0], mesh->vertexPoints[tet.v1], mesh->vertexPoints[tet.v2], mesh->vertexPoints[tet.v3],
				mesh->vertexPoints[tet.v4], mesh->vertexPoints[tet.v5], mesh->vertexPoints[tet.v6], mesh->vertexPoints[tet.v7] };
			float q = qualityHex(points, q_crit);
			myAtomicMin(&q_min[0], q);
		}
	}

	// checks for non decreasing quality of every element
	__global__ static void k_updateCurrentQualities(MeshHexDevice* mesh, ArrayView<float> currentQualities, float lastMinQuality, ArrayView<bool> failure, QualityCriterium q_crit) {
		for (int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < mesh->nHexahedra; idx += blockDim.x * gridDim.x) {
			const Hexahedron& tet = mesh->hexahedra[idx];
			const Vec3f points[8] = { mesh->vertexPoints[tet.v0], mesh->vertexPoints[tet.v1], mesh->vertexPoints[tet.v2], mesh->vertexPoints[tet.v3],
				mesh->vertexPoints[tet.v4], mesh->vertexPoints[tet.v5], mesh->vertexPoints[tet.v6], mesh->vertexPoints[tet.v7] };
			float q = qualityHex(points, q_crit);
			if (q < lastMinQuality) {
				failure[0] = true;
				printf("minimum quality decreased! hex %i vertices %i %i %i %i %i %i %i %i q %f\n", idx, tet.v0, tet.v1, tet.v2, tet.v3, tet.v4, tet.v5, tet.v6, tet.v7, q);
				assert(0);
			}
			currentQualities[idx] = q;
		}
	}

	__global__ static void k_getElementQualities(MeshHexDevice* mesh, float* outBuffer, QualityCriterium q_crit) {
		for (int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < mesh->nHexahedra; idx += blockDim.x * gridDim.x) {
			const Hexahedron& tet = mesh->hexahedra[idx];
			const Vec3f points[8] = { mesh->vertexPoints[tet.v0], mesh->vertexPoints[tet.v1], mesh->vertexPoints[tet.v2], mesh->vertexPoints[tet.v3],
				mesh->vertexPoints[tet.v4], mesh->vertexPoints[tet.v5], mesh->vertexPoints[tet.v6], mesh->vertexPoints[tet.v7] };
			outBuffer[idx] = qualityHex(points, q_crit);
		}
	}


	// ######################################################################## //
	// ### DMOTetClass ######################################################## //
	// ######################################################################## //
	
	DMOHexClass::DMOHexClass(DMOMeshHex& dmo_mesh_, QualityCriterium qualityCriterium_, const float gridScale_, int n_iter_)
		: DMOBaseClass(qualityCriterium_, gridScale_, n_iter_)
		, dmo_mesh(dmo_mesh_)
		//, qualityCriterium(qualityCriterium_)
		//, gridScale(gridScale_)
		//, n_iter(n_iter_)
	{
		init();
	}

	void DMOHexClass::init() {
		vertexPointsInit = device_vector<Vec3f>(dmo_mesh.vertexPoints, dmo_mesh.vertexPoints + dmo_mesh.nVertices); // copy of initial vertex positions

		localSurfacesInit1d = device_vector<localSurface1d>(dmo_mesh.nVerticesSurf); // initial local surfaces1d
		computeLocalSurfaces1d(dmo_mesh, localSurfacesInit1d);

		localSurfacesInit = device_vector<localSurface>(dmo_mesh.nVerticesSurf); // initial local surfaces
		computeLocalSurfaces(dmo_mesh, localSurfacesInit);

		// new
		if (dmo_mesh.nVerticesFeature > 0)
			initLocalSurfacesFeature(dmo_mesh, surfacesRowPtr, localSurfacesFeatureInit, table);

		nearestNeighbors = device_vector<int>(dmo_mesh.nVerticesSurf);
		thrust::sequence(nearestNeighbors.begin(), nearestNeighbors.end(), 0);

		optimizeFeatureVertex = device_vector<bool>(dmo_mesh.nVerticesSurf, true);
		calcOptFeatureVec(dmo_mesh, optimizeFeatureVertex);

		lastMinQuality = findMinimumQuality();

		cout << "Total points " << dmo_mesh.nVertices << endl;
		cout << "Inner points " << dmo_mesh.nVertices - dmo_mesh.nVerticesSurf << endl;
		cout << "Free Surf points " << dmo_mesh.nVerticesSurf - dmo_mesh.nVerticesFeature << endl;
		cout << "Feature Surf points " << dmo_mesh.nVerticesFeature << endl;

		cout << "Colors Free " << dmo_mesh.nColorsFree << endl;
		cout << "Colors Feature " << dmo_mesh.nColorsFeature << endl;
		cout << "Colors Inner " << dmo_mesh.nColorsInner << endl;

		currentQualities = device_vector<float>(dmo_mesh.nHexahedra, -FLT_MAX);

		cout << "0," << lastMinQuality << endl;
	}


	void DMOHexClass::doIteration() {
		int dynMemSize3D = dmo_mesh.maxNumHexahedra * 7 * 3 * sizeof(float);
		int dynMemSize2D = dmo_mesh.maxNumHexahedra * 7 * 3 * sizeof(float) + dmo_mesh.maxNumHalfedges * sizeof(int);
		int dynMemSize1D = dmo_mesh.maxNumHexahedra * 7 * 3 * sizeof(float) + dmo_mesh.maxNumHalfedges * sizeof(int);

		for (int cid = 0; cid < dmo_mesh.nColorsInner; ++cid) {
			//for (int i = dmo_mesh.colorOffsetsInner[cid]; i < dmo_mesh.colorOffsetsInner[cid + 1]; ++i) {
			//	cout << "opt vid " << i << endl;
			//	k_optimizeHierarchicalInnerHex<128> << <1, 128, dynMemSize3D >> >
			//		(i, i+1, dmo_mesh.d_mesh, affineFactor, qualityCriterium, gridScale);
			//	gpuErrchk(hipPeekAtLastError());
			//	gpuErrchk(hipDeviceSynchronize());
			//	updateCurrentQualities();
			//}

			k_optimizeHierarchicalInnerHex<128> << <dmo_mesh.colorOffsetsInner[cid + 1] - dmo_mesh.colorOffsetsInner[cid], 128, dynMemSize3D >> >
				(dmo_mesh.colorOffsetsInner[cid], dmo_mesh.colorOffsetsInner[cid + 1], dmo_mesh.d_mesh, affineFactor, qualityCriterium, gridScale);
			gpuErrchk(hipPeekAtLastError());
			gpuErrchk(hipDeviceSynchronize());
		}

		if (lastMinQuality > 0.f) {
			for (int cid = 0; cid < dmo_mesh.nColorsFree; ++cid) {
				k_optimizeHierarchical2DHex<DMO_NQ * DMO_NQ/2, USE_SURF_OF_NN> << <dmo_mesh.colorOffsetsFree[cid + 1] - dmo_mesh.colorOffsetsFree[cid], DMO_NQ* DMO_NQ/2, dynMemSize2D >> >
					(dmo_mesh.colorOffsetsFree[cid], dmo_mesh.colorOffsetsFree[cid + 1], dmo_mesh.d_mesh, affineFactor, qualityCriterium, gridScale,
						raw_pointer_cast(vertexPointsInit.data()), raw_pointer_cast(localSurfacesInit.data()), raw_pointer_cast(nearestNeighbors.data()),
						raw(table), raw(surfacesRowPtr), raw(localSurfacesFeatureInit), dmo_mesh.maxNumHexahedra * 7);
				gpuErrchk(hipPeekAtLastError());
				gpuErrchk(hipDeviceSynchronize());
			}

			for (int cid = 0; cid < dmo_mesh.nColorsFeature; ++cid) {
				k_optimizeHierarchical1DHex<DMO_NQ> << <dmo_mesh.colorOffsetsFeature[cid + 1] - dmo_mesh.colorOffsetsFeature[cid], DMO_NQ, dynMemSize1D >> >
					(dmo_mesh.colorOffsetsFeature[cid], dmo_mesh.colorOffsetsFeature[cid + 1], dmo_mesh.d_mesh, affineFactor, qualityCriterium, gridScale,
						raw_pointer_cast(vertexPointsInit.data()), raw_pointer_cast(localSurfacesInit1d.data()), raw_pointer_cast(nearestNeighbors.data()),
						raw_pointer_cast(optimizeFeatureVertex.data()), dmo_mesh.maxNumHexahedra * 7, dmo_mesh.maxNumHalfedges);
				gpuErrchk(hipPeekAtLastError());
				gpuErrchk(hipDeviceSynchronize());
			}
		}

		updateNearestNeighbor(dmo_mesh, vertexPointsInit, nearestNeighbors);
		dmo_mesh.updateNormals();
		++curr_it;
		
		updateCurrentQualities();
		float newQuality = findMinimumQuality();
		//cout << "new quality: " << newQuality << endl;
		cout << curr_it << "," << newQuality << endl;
		assert(newQuality >= lastMinQuality);
		lastMinQuality = newQuality;
	}

	void DMOHexClass::getEstimateLocalSurfacePoints(int vid, int nu, int nv, void* outSurfacePoints) {
		int dynMemSize = dmo_mesh.maxNumHalfedges * sizeof(int);
		k_fillEstimateLocalSurfacePoints<USE_SURF_OF_NN> << <1, nu* nv, dynMemSize>> > (vid, nu, nv, dmo_mesh.d_mesh, affineFactor, gridScale,
			raw_pointer_cast(vertexPointsInit.data()), raw_pointer_cast(localSurfacesInit.data()), raw_pointer_cast(nearestNeighbors.data()), (Vec3f*)outSurfacePoints,
			raw(table), raw(surfacesRowPtr), raw(localSurfacesFeatureInit));
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
	}

	void DMOHexClass::getLocalSurfacePoints(int vid, int nu, int nv, void* outSurfacePoints, int featureSid) {
		k_fillLocalSurfacePoints << <1, nu* nv >> > (vid, featureSid, nu, nv, dmo_mesh.d_mesh, affineFactor, gridScale,
			raw(localSurfacesInit), raw(nearestNeighbors), (Vec3f*)outSurfacePoints, raw(localSurfacesFeatureInit), raw(surfacesRowPtr));
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
	}

	void DMOHexClass::displayQualityGPU(int n_cols) {
		device_vector<int> q_vec(n_cols, 0);
		device_vector<float> q_min(1, FLT_MAX);
		const int BLOCK_SIZE = 128;
		k_qualityHistogram << <getBlockCount(dmo_mesh.nHexahedra, BLOCK_SIZE), BLOCK_SIZE >> > (dmo_mesh.d_mesh, q_vec, q_min, n_cols, qualityCriterium);
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());

		host_vector<int> q_vecHost(q_vec);
		host_vector<float> q_minHost(q_min);

		printFormattedQuality(q_vecHost, q_minHost);
	}

	void DMOHexClass::getQualityHistogram(std::vector<int>& vec, int n_cols) {
		device_vector<int> q_vec(n_cols, 0);
		device_vector<float> q_min(1, FLT_MAX);
		const int BLOCK_SIZE = 128;
		k_qualityHistogram << <getBlockCount(dmo_mesh.nHexahedra, BLOCK_SIZE), BLOCK_SIZE >> > (dmo_mesh.d_mesh, q_vec, q_min, n_cols, qualityCriterium);
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());

		host_vector<int> q_vecHost(q_vec);
		thrust::copy(q_vec.begin(), q_vec.end(), vec.begin());
	}

	float DMOHexClass::findMinimumQuality() const {
		device_vector<float> q_min(1, FLT_MAX);
		const int BLOCK_SIZE = 128;
		k_findMinimumQuality << <getBlockCount(dmo_mesh.nHexahedra, BLOCK_SIZE), BLOCK_SIZE >> > (dmo_mesh.d_mesh, q_min, qualityCriterium);
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
		return q_min[0];
	}

	void DMOHexClass::updateCurrentQualities() {
		const int BLOCK_SIZE = 128;
		device_vector<bool> failure(1, false);
		k_updateCurrentQualities << <getBlockCount(dmo_mesh.nHexahedra, BLOCK_SIZE), BLOCK_SIZE >> > (dmo_mesh.d_mesh, currentQualities, lastMinQuality, failure, qualityCriterium);
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
		if (failure[0]) {
			writeHex("res/fail_debug", dmo_mesh);
			throw 1;
		}
	}

	void DMOHexClass::getElementQualities(void* outFloatBuffer) {
		const int BLOCK_SIZE = 128;
		k_getElementQualities << <getBlockCount(dmo_mesh.nHexahedra, BLOCK_SIZE), BLOCK_SIZE >> > (dmo_mesh.d_mesh, (float*)outFloatBuffer, qualityCriterium);
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
	}
	
}


