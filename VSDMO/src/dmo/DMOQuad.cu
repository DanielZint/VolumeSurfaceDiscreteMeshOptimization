#include "hip/hip_runtime.h"
#include "DMOQuad.h"

#include "hip/hip_runtime.h"
#include ""
#include <thrust/count.h>
#include <thrust/sequence.h>
#include "CudaUtil.h"
#include "CudaAtomic.h"
#include "DMOCommon.h"
#include "SurfaceConfig.h"

#include "io/FileWriter.h"
#include "Serializer.h"

#include "DMOQuadImplCub.h"


#ifdef USECUB
using namespace DMOImplCub;
#else
using namespace DMOImplAtomic;
#endif

namespace DMO {
	

	// counts the number of elements with quality between 0 - 0.1 and so on.
	__global__ static void k_qualityHistogram(MeshQuadDevice* mesh, ArrayView<int> q_vec, ArrayView<float> q_min, int n_cols, QualityCriterium q_crit) {
		for (int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < mesh->nQuads; idx += blockDim.x * gridDim.x) {
			const Quad& quad = mesh->quads[idx];
			const Vec3f points[4] = { mesh->vertexPoints[quad.v0], mesh->vertexPoints[quad.v1], mesh->vertexPoints[quad.v2], mesh->vertexPoints[quad.v3] };
			float q = qualityQuad(points, q_crit); // TODO qualityQuad
			myAtomicMin(&q_min[0], q);

			q = fminf(0.9999f, q);
			size_t index = size_t(q * n_cols);
			atomicAdd(&q_vec[index], 1);
		}
	}

	// finds the minimum element quality
	__global__ static void k_findMinimumQuality(MeshQuadDevice* mesh, ArrayView<float> q_min, QualityCriterium q_crit) {
		for (int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < mesh->nQuads; idx += blockDim.x * gridDim.x) {
			const Quad& quad = mesh->quads[idx];
			const Vec3f points[4] = { mesh->vertexPoints[quad.v0], mesh->vertexPoints[quad.v1], mesh->vertexPoints[quad.v2], mesh->vertexPoints[quad.v3] };
			float q = qualityQuad(points, q_crit);
			myAtomicMin(&q_min[0], q);
		}
	}

	// checks for non decreasing quality of every element
	__global__ static void k_updateCurrentQualities(MeshQuadDevice* mesh, ArrayView<float> currentQualities, float lastMinQuality, ArrayView<bool> failure, QualityCriterium q_crit) {
		for (int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < mesh->nQuads; idx += blockDim.x * gridDim.x) {
			const Quad& quad = mesh->quads[idx];
			const Vec3f points[4] = { mesh->vertexPoints[quad.v0], mesh->vertexPoints[quad.v1], mesh->vertexPoints[quad.v2], mesh->vertexPoints[quad.v3] };
			float q = qualityQuad(points, q_crit);
			if (q < lastMinQuality) {
				failure[0] = true;
				printf("minimum quality decreased! quad %i vertices %i %i %i q %f\n", idx, quad.v0, quad.v1, quad.v2, q);
				assert(0);
			}
			currentQualities[idx] = q;
		}
	}

	__global__ static void k_getElementQualities(MeshQuadDevice* mesh, float* outBuffer, QualityCriterium q_crit) {
		for (int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < mesh->nQuads; idx += blockDim.x * gridDim.x) {
			const Quad& quad = mesh->quads[idx];
			const Vec3f points[4] = { mesh->vertexPoints[quad.v0], mesh->vertexPoints[quad.v1], mesh->vertexPoints[quad.v2], mesh->vertexPoints[quad.v3] };
			outBuffer[idx] = qualityQuad(points, q_crit);
		}
	}


	
	// ######################################################################## //
	// ### DMOTriClass ######################################################## //
	// ######################################################################## //

	DMOQuadClass::DMOQuadClass(DMOMeshQuad& dmo_mesh_, QualityCriterium qualityCriterium_, const float gridScale_, int n_iter_)
		: DMOBaseClass(qualityCriterium_, gridScale_, n_iter_)
		, dmo_mesh(dmo_mesh_)
		//, qualityCriterium(qualityCriterium_)
		//, gridScale(gridScale_)
		//, n_iter(n_iter_)
	{
		init();
	}

	void DMOQuadClass::init() {
		vertexPointsInit = device_vector<Vec3f>(dmo_mesh.vertexPoints, dmo_mesh.vertexPoints + dmo_mesh.nVerticesSurf); // copy of initial vertex positions

		localSurfacesInit1d = device_vector<localSurface1d>(dmo_mesh.nVerticesSurf); // initial local surfaces1d
		computeLocalSurfaces1d(dmo_mesh, localSurfacesInit1d);

		localSurfacesInit = device_vector<localSurface>(dmo_mesh.nVerticesSurf); // initial local surfaces
		computeLocalSurfaces(dmo_mesh, localSurfacesInit);

		// new
		if (dmo_mesh.nVerticesFeature > 0)
			initLocalSurfacesFeature(dmo_mesh, surfacesRowPtr, localSurfacesFeatureInit, table);

		//writeSurfaces(localSurfacesInit, "res/surfaces/surfaces2d2.binary");

		nearestNeighbors = device_vector<int>(dmo_mesh.nVerticesSurf);
		thrust::sequence(nearestNeighbors.begin(), nearestNeighbors.end(), 0);

		optimizeFeatureVertex = device_vector<bool>(dmo_mesh.nVerticesSurf, true);
		calcOptFeatureVec(dmo_mesh, optimizeFeatureVertex);

		lastMinQuality = findMinimumQuality();

		cout << "Total points " << dmo_mesh.nVerticesSurf << endl;
		cout << "Free Surf points " << dmo_mesh.nVerticesSurf - dmo_mesh.nVerticesFeature << endl;
		cout << "Feature Surf points " << dmo_mesh.nVerticesFeature << endl;

		cout << "Colors Free " << dmo_mesh.nColorsFree << endl;
		cout << "Colors Feature " << dmo_mesh.nColorsFeature << endl;

		currentQualities = device_vector<float>(dmo_mesh.nQuads, -FLT_MAX);

		cout << "0," << lastMinQuality << endl;
	}


	void DMOQuadClass::doIteration() {
		int dynMemSize2D = (2 * dmo_mesh.maxNumHalfedges + 1) * 3 * sizeof(float) + dmo_mesh.maxNumHalfedges * sizeof(int);
		int dynMemSize1D = (2 * dmo_mesh.maxNumHalfedges + 1) * 3 * sizeof(float) + dmo_mesh.maxNumHalfedges * sizeof(int);

		for (int cid = 0; cid < dmo_mesh.nColorsFree; ++cid) {
			//cout << "color offsets " << (dmo_mesh.colorOffsetsFree[cid + 1] - dmo_mesh.colorOffsetsFree[cid]) << endl;
			//cout << dynMemSize2D << endl;
			k_optimizeHierarchical2D<DMO_NQ* DMO_NQ/2, USE_SURF_OF_NN> << <dmo_mesh.colorOffsetsFree[cid + 1] - dmo_mesh.colorOffsetsFree[cid], DMO_NQ* DMO_NQ/2, dynMemSize2D >> >
				(dmo_mesh.colorOffsetsFree[cid], dmo_mesh.colorOffsetsFree[cid + 1], dmo_mesh.d_mesh, affineFactor, qualityCriterium, gridScale,
					raw_pointer_cast(vertexPointsInit.data()), raw_pointer_cast(localSurfacesInit.data()), raw_pointer_cast(nearestNeighbors.data()),
					raw(table), raw(surfacesRowPtr), raw(localSurfacesFeatureInit), 2 * dmo_mesh.maxNumHalfedges + 1);
			gpuErrchk(hipPeekAtLastError());
			gpuErrchk(hipDeviceSynchronize());
		}

		for (int cid = 0; cid < dmo_mesh.nColorsFeature; ++cid) {
			k_optimizeHierarchical1D<DMO_NQ> << <dmo_mesh.colorOffsetsFeature[cid + 1] - dmo_mesh.colorOffsetsFeature[cid], DMO_NQ, dynMemSize1D >> >
				(dmo_mesh.colorOffsetsFeature[cid], dmo_mesh.colorOffsetsFeature[cid + 1], dmo_mesh.d_mesh, affineFactor, qualityCriterium, gridScale,
					raw_pointer_cast(vertexPointsInit.data()), raw_pointer_cast(localSurfacesInit1d.data()), raw_pointer_cast(nearestNeighbors.data()),
					raw_pointer_cast(optimizeFeatureVertex.data()), 2 * dmo_mesh.maxNumHalfedges + 1, dmo_mesh.maxNumHalfedges);
			gpuErrchk(hipPeekAtLastError());
			gpuErrchk(hipDeviceSynchronize());
		}


		updateNearestNeighbor(dmo_mesh, vertexPointsInit, nearestNeighbors);
		dmo_mesh.updateNormals();
		++curr_it;

		updateCurrentQualities();
		float newQuality = findMinimumQuality();
		//cout << "new quality: " << newQuality << endl;
		cout << curr_it << "," << newQuality << endl;
		assert(newQuality >= lastMinQuality);
		lastMinQuality = newQuality;
	}





	void DMOQuadClass::getEstimateLocalSurfacePoints(int vid, int nu, int nv, void* outSurfacePoints) {
		int dynMemSize = dmo_mesh.maxNumHalfedges * sizeof(int);
		k_fillEstimateLocalSurfacePoints<USE_SURF_OF_NN> << <1, nu* nv, dynMemSize>> > (vid, nu, nv, dmo_mesh.d_mesh, affineFactor, gridScale,
			raw_pointer_cast(vertexPointsInit.data()), raw_pointer_cast(localSurfacesInit.data()), raw_pointer_cast(nearestNeighbors.data()), (Vec3f*)outSurfacePoints,
			raw(table), raw(surfacesRowPtr), raw(localSurfacesFeatureInit));
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
	}

	void DMOQuadClass::getLocalSurfacePoints(int vid, int nu, int nv, void* outSurfacePoints, int featureSid) {
		k_fillLocalSurfacePoints << <1, nu* nv >> > (vid, featureSid, nu, nv, dmo_mesh.d_mesh, affineFactor, gridScale,
			raw(localSurfacesInit), raw(nearestNeighbors), (Vec3f*)outSurfacePoints, raw(localSurfacesFeatureInit), raw(surfacesRowPtr));
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
	}

	void DMOQuadClass::displayQualityGPU(int n_cols) {
		device_vector<int> q_vec(n_cols, 0);
		device_vector<float> q_min(1, FLT_MAX);
		const int BLOCK_SIZE = 128;
		k_qualityHistogram << <getBlockCount(dmo_mesh.nQuads, BLOCK_SIZE), BLOCK_SIZE >> > (dmo_mesh.d_mesh, q_vec, q_min, n_cols, qualityCriterium);
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());

		host_vector<int> q_vecHost(q_vec);
		host_vector<float> q_minHost(q_min);

		printFormattedQuality(q_vecHost, q_minHost);
	}

	void DMOQuadClass::getQualityHistogram(std::vector<int>& vec, int n_cols) {
		device_vector<int> q_vec(n_cols, 0);
		device_vector<float> q_min(1, FLT_MAX);
		const int BLOCK_SIZE = 128;
		k_qualityHistogram << <getBlockCount(dmo_mesh.nQuads, BLOCK_SIZE), BLOCK_SIZE >> > (dmo_mesh.d_mesh, q_vec, q_min, n_cols, qualityCriterium);
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());

		host_vector<int> q_vecHost(q_vec);
		thrust::copy(q_vec.begin(), q_vec.end(), vec.begin());
	}

	float DMOQuadClass::findMinimumQuality() const {
		device_vector<float> q_min(1, FLT_MAX);
		const int BLOCK_SIZE = 128;
		k_findMinimumQuality << <getBlockCount(dmo_mesh.nQuads, BLOCK_SIZE), BLOCK_SIZE >> > (dmo_mesh.d_mesh, q_min, qualityCriterium);
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
		return q_min[0];
	}

	void DMOQuadClass::updateCurrentQualities() {
		const int BLOCK_SIZE = 128;
		device_vector<bool> failure(1, false);
		k_updateCurrentQualities << <getBlockCount(dmo_mesh.nQuads, BLOCK_SIZE), BLOCK_SIZE >> > (dmo_mesh.d_mesh, currentQualities, lastMinQuality, failure, qualityCriterium);
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
		if (failure[0]) {
			writeOFF("res/fail_debug", dmo_mesh);
			throw 1;
		}
	}

	void DMOQuadClass::getElementQualities(void* outFloatBuffer) {
		const int BLOCK_SIZE = 128;
		k_getElementQualities << <getBlockCount(dmo_mesh.nQuads, BLOCK_SIZE), BLOCK_SIZE >> > (dmo_mesh.d_mesh, (float*)outFloatBuffer, qualityCriterium);
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
	}

}


